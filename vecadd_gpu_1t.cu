/*
Vector Addition code taken from Mark Harris: 

https://developer.nvidia.com/blog/even-easier-introduction-cuda/

Modified problem size to N=1<<26 (64M) 
*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)  //global runs on gpu
{
  for (int i = 0; i < n; i++)
      y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<26; // 64M elements
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));


 // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  /* timer code here taken from Benchmark.cpp
  std::chrono::time_point<std::chrono::high_resolution_clock> start_time = std::chrono::high_resolution_clock::now();
  */

  // Run kernel on 1M elements on the CPU
  add<<<1, 1>>>(N, x, y);

  /* timer code here taken from Benchmark.cpp
  std::chrono::time_point<std::chrono::high_resolution_clock> end_time = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = end_time - start_time;
  std::cout << "Elapsed time is: " << elapsed.count() << " seconds" << std::endl;
  */

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;
  
 // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}